#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cudf/utilities/error.hpp>
#include "GPUManager.cuh"

namespace ral {
namespace config {

size_t gpuFreeMemory() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return free;
}

size_t gpuTotalMemory() {
	int nDevices;

	hipGetDeviceCount(&nDevices);

	size_t total_memory;
	size_t free, total;
	struct hipDeviceProp_t props;
	for (int gpu_id = 0; gpu_id < nDevices; ++gpu_id) {
		CUDA_TRY( hipSetDevice(gpu_id) );
		hipGetDeviceProperties(&props, gpu_id);
		hipMemGetInfo(&free, &total);
		total_memory += total;
	}
	return total_memory;
}

size_t gpuUsedMemory() {
	int currentDeviceId = 0;
	struct hipDeviceProp_t props;
	CUDA_TRY( hipSetDevice(currentDeviceId) );
	hipGetDeviceProperties(&props, currentDeviceId);
	size_t free, total;
	hipMemGetInfo(&free, &total);

	return total - free;
}

}	// namespace config
}	// namespace ral
